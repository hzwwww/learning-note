#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                         \
{                                                                           \
    const mcError_t error = call;                                           \
    if (err != hipSuccess) {                                               \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1);                                                            \
    }                                                                       \
}                                                                           \

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i=0; i<N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Arrays do not match\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match. \n\n");
}

void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned) time(&t));

    for (int i=0; i<size; i++) {
        ip[i] = (float) (rand() & 0xFF)/10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int i=0; i<N; i++) {
        C[i] = A[i] + B[i];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
    // 使用线程id建立线程与数据的映射
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
    printf("%s Starting...\n", argv[0]);

    // 绑定device
    int dev = 0;
    mcSetDevice(dev);

    // 设置向量大小
    int nElem = 32;
    printf("Vector size %d\n", nElem);

    // 申请host内存
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes); // 保存host的计算结果
    gpuRef = (float *)malloc(nBytes); // 保存device的计算结果（从device拷贝）

    // 初始化数据
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    // 数据置0，保证状态正确，防止出现脏数据，便于调试（0代表数据未填充）
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // 申请device全局内存
    float *d_A, *d_B, *d_C;
    mcMalloc(&d_A, nBytes);
    mcMalloc(&d_B, nBytes);
    mcMalloc(&d_C, nBytes);

    // 复制host数据到device
    mcMemcpy(d_A, h_A, nBytes, mcMemcpyHostToDevice);
    mcMemcpy(d_B, h_B, nBytes, mcMemcpyHostToDevice);

    // 在host端调用kernel计算向量和
    dim3 block (nElem); // 单维block，每个block处理nElem个元素
    dim3 grid (nElem/block.x); // 根据每个block处理的元素数量，计算grid所需的block数

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d, %d>>>\n", grid.x, block.x);

    // 复制device数据到host
    mcMemcpy(gpuRef, d_C, nBytes, mcMemcpyDeviceToHost);

    // 在host端计算向量和
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // 检查device计算结果
    checkResult(hostRef, gpuRef, nElem);

    // 释放device全局内存
    mcFree(d_A);
    mcFree(d_B);
    mcFree(d_C);

    // 释放host内存
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return(0);
}