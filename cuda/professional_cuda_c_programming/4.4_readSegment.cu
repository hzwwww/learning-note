#include "hip/hip_runtime.h"
#include "./common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This example demonstrates the impact of misaligned reads on performance by
 * forcing misaligned reads to occur on a float*.
 */

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("different on %dth element: host %f gpu %f\n", i, hostRef[i],
                    gpuRef[i]);
            break;
        }
    }

    if (!match)  printf("Arrays do not match.\n\n");
}

void initialData(float *ip,  int size)
{
    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 100.0f;
    }

    return;
}


void sumArraysOnHost(float *A, float *B, float *C, const int n, int offset)
{
    for (int idx = offset, k = 0; idx < n; idx++, k++)
    {
        C[k] = A[idx] + B[idx];
    }
}

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

__global__ void readOffset(float *A, float *B, float *C, const int n,
                           int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) C[i] = A[k] + B[k];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    mcDeviceProp_t deviceProp;
    CHECK(mcGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(mcSetDevice(dev));

    // set up array size
    int nElem = 1 << 20; // total number of elements to reduce
    printf(" with array size %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);

    // set up offset for summary
    int blocksize = 512;
    int offset = 0;

    if (argc > 1) offset    = atoi(argv[1]);

    if (argc > 2) blocksize = atoi(argv[2]);

    // execution configuration
    dim3 block (blocksize, 1);
    dim3 grid  ((nElem + block.x - 1) / block.x, 1);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef  = (float *)malloc(nBytes);

    //  initialize host array
    initialData(h_A, nElem);
    memcpy(h_B, h_A, nBytes);

    //  summary at host side
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);

    // allocate device memory
    float *d_A, *d_B, *d_C;
    CHECK(mcMalloc((float**)&d_A, nBytes));
    CHECK(mcMalloc((float**)&d_B, nBytes));
    CHECK(mcMalloc((float**)&d_C, nBytes));

    // copy data from host to device
    CHECK(mcMemcpy(d_A, h_A, nBytes, mcMemcpyHostToDevice));
    CHECK(mcMemcpy(d_B, h_A, nBytes, mcMemcpyHostToDevice));

    //  kernel 1:
    double iStart = seconds();
    warmup<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(mcDeviceSynchronize());
    double iElaps = seconds() - iStart;
    printf("warmup     <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CHECK(mcGetLastError());

    iStart = seconds();
    readOffset<<<grid, block>>>(d_A, d_B, d_C, nElem, offset);
    CHECK(mcDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("readOffset <<< %4d, %4d >>> offset %4d elapsed %f sec\n", grid.x,
           block.x, offset, iElaps);
    CHECK(mcGetLastError());

    // copy kernel result back to host side and check device results
    CHECK(mcMemcpy(gpuRef, d_C, nBytes, mcMemcpyDeviceToHost));
    checkResult(hostRef, gpuRef, nElem - offset);

    // free host and device memory
    CHECK(mcFree(d_A));
    CHECK(mcFree(d_B));
    CHECK(mcFree(d_C));
    free(h_A);
    free(h_B);

    // reset device
    CHECK(mcDeviceReset());
    return EXIT_SUCCESS;
}